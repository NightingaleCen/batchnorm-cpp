#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include "sharedmem.cuh"

#include <vector>

namespace {

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_forward_mean_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> mu_cache) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      mu_cache[n][i] = input[n][i] / batch_size;
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_forward_sigma2_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> sigma2_cache
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      sigma2_cache[n][i] = (input[n][i] - mu[i]) * (input[n][i] - mu[i]) / batch_size;
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_forward_output_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> normalized_input,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> beta,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      normalized_input[n][i] = (input[n][i] - mu[i]) * rsqrt(sigma2[i] + 1e-5);
      output[n][i] = gamma[i] * normalized_input[n][i] + beta[i];
    }
  }

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_normalized_input_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_normalized_input
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_normalized_input[n][i] = d_output[n][i] * gamma[i];
      //////printf("channel: %d\nd_normalized_input: %f,d_output: %f, gamma: %f\n", i, d_normalized_input[n][i][r][c], d_output[n][i][r][c], gamma[i]);
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_gamma_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> normalized_input,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_gamma_cache
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_gamma_cache[n][i] = d_output[n][i] * normalized_input[n][i];
      //////printf("d_gamma_cache: %f\n", d_gamma_cache[n][i][r][c]);

    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_beta_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_beta_cache
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_beta_cache[n][i] = d_output[n][i];
      //printf("d_beta_cache: %f\n", d_beta_cache[n][i][r][c]);
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_sigma2_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_sigma2_cache) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_sigma2_cache[n][i] = d_output[n][i] * (input[n][i] - mu[i]) * ((gamma[i] / (-2)) * rsqrt(sigma2[i] + 1e-5) * rsqrt(sigma2[i] + 1e-5) * rsqrt(sigma2[i] + 1e-5));
      //////printf("d_sigma2_cache: %f,d_output: %f,input: %f,mu: %f,sigma2: %f,gamma: %f\n", d_sigma2_cache[n][i][r][c], d_output[n][i][r][c], input[n][i][r][c], mu[i], sigma2[i], gamma[i]);
    }
  }

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_mu_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_sigma2,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_mu_cache) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_mu_cache[n][i] = (((-2) * d_sigma2[i] * (input[n][i] - mu[i])) / batch_size) + (d_output[n][i] * (-1) * gamma[i] * rsqrt(sigma2[i] + 1e-5));
      //////printf("d_mu_cache: %f,d_sigma2: %f,d_output: %f,input: %f,mu: %f,sigma2: %f,gamma: %f,slice_size: %u, rsqrt: %f\n", d_mu_cache[n][i][r][c], d_sigma2[i], d_output[n][i][r][c], input[n][i][r][c], mu[i], sigma2[i], gamma[i], slice_size, rsqrt(sigma2[i] + 1e-5));
    }
  }

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_input_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> normalized_input,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_normalized_input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_mu,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_input[n][i] = (d_normalized_input[n][i] * rsqrt(sigma2[i] + 1e-5)) + (d_sigma2[i] * ((2 * (input[n][i] - mu[i])) / batch_size) + (d_mu[i] / batch_size));
      //////printf("d_input: %f,d_mu: %f,d_sigma2: %f,input: %f,d_normalized_input: %f,mu: %f,sigma2: %f,slice_size: %u\n", d_input[n][i][r][c], d_mu[i], d_sigma2[i], input[n][i][r][c], d_normalized_input[n][i][r][c], mu[i], sigma2[i], slice_size);
    }
  }
} // namespace

template <typename scalar_t>
__global__ void reduction_sum1d_kernel(scalar_t* array, scalar_t* sum_output, const int64_t total_num) {

  SharedMemory<scalar_t> shared_memory;
  scalar_t* shared_data = shared_memory.getPointer();
  const auto tid = threadIdx.x;
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;


  if (i < total_num) {
    shared_data[tid] = array[i];
    //printf("%f ,", array[i]);
  }
  else {
    shared_data[tid] = 0;
  }

  __syncthreads();

  for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      shared_data[tid] += shared_data[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    //printf("\n");
    sum_output[blockIdx.x] = shared_data[0];
  }
}

template <typename scalar_t>
scalar_t reduction_sum(
  torch::Tensor array
) {

  const auto total_num = array.numel();
  const auto thread_num = 1024;
  const auto block_num = (total_num + 1024 - 1) / 1024;

  scalar_t* block_sums; // block_sum[-1] represents the total sum of array
  hipMalloc((void**)&block_sums, sizeof(scalar_t) * (1 + block_num));
  //printf("Block num: %d\n", block_num);
  //printf("Redution total num: %d\n", total_num);

  reduction_sum1d_kernel<scalar_t> << < block_num, thread_num, thread_num * sizeof(scalar_t) >> > (array.data_ptr<scalar_t>(), block_sums, total_num);
  hipDeviceSynchronize();
  reduction_sum1d_kernel<scalar_t> << <1, thread_num, thread_num * sizeof(scalar_t) >> > (block_sums, block_sums + block_num, block_num);
  hipDeviceSynchronize();


  scalar_t total_sum;
  hipMemcpy(&total_sum, block_sums + block_num, sizeof(scalar_t), hipMemcpyDeviceToHost);
  hipFree(block_sums);

  //printf("Redution total sum: %f\n", total_sum);

  return total_sum;

}

std::vector<torch::Tensor> batchnorm1d_cuda_forward(
  torch::Tensor input,
  torch::Tensor gamma,
  torch::Tensor beta) {
  const auto batch_size = input.size(0);
  const auto C = input.size(1);

  const int thread_batch_num = int(floor(sqrt(1024 / C)));

  auto mu = torch::zeros_like(gamma);
  auto sigma2 = torch::zeros_like(gamma);
  auto normalized_input = torch::zeros_like(input);
  auto output = torch::zeros_like(input);
  auto cache = torch::empty_like(input);

  const dim3 threads(thread_batch_num, C, 1);
  const dim3 blocks((batch_size + thread_batch_num - 1) / thread_batch_num, 1, 1);

  //printf("Forward:\n");
  //printf("Dim: %d, %d, %d, %d\n", batch_size, C, H, W);
  //printf("DimSize: %d\n", dim_size);
  //printf("blockDim: %d, %d, %d\n", threads.x, threads.y, threads.z);
  //printf("gridDim: %d, %d, %d\n", blocks.x, blocks.y, blocks.z);


  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "batchnorm1d_forward_cuda", ([&] {
    batchnorm1d_cuda_forward_mean_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
    input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
    cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    mu.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_forward_sigma2_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                             input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    sigma2.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_forward_output_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                             input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             beta.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
  hipDeviceSynchronize();
                                                                               }));
  return { input, mu, sigma2, normalized_input, gamma, beta, output };
}

std::vector<torch::Tensor> batchnorm1d_cuda_backward(
  torch::Tensor d_output,
  torch::Tensor input,
  torch::Tensor mu,
  torch::Tensor sigma2,
  torch::Tensor normalized_input,
  torch::Tensor gamma,
  torch::Tensor beta) {
  const auto batch_size = input.size(0);
  const auto C = input.size(1);

  const int thread_batch_num = int(floor(sqrt(1024 / C)));

  auto cache = torch::empty_like(input);

  auto d_input = torch::empty_like(input);
  auto d_gamma = torch::empty_like(gamma);
  auto d_beta = torch::empty_like(beta);

  auto d_normalized_input = torch::empty_like(normalized_input);
  auto d_sigma2 = torch::empty_like(sigma2);
  auto d_mu = torch::empty_like(mu);

  const dim3 threads(thread_batch_num, C, 1);
  const dim3 blocks((batch_size + thread_batch_num - 1) / thread_batch_num, 1, 1);

  //printf("Backward:\n");
  //printf("Dim: %d, %d, %d, %d\n", batch_size, C, H, W);
  //printf("DimSize: %d\n", dim_size);
  //printf("blockDim: %d, %d, %d\n", threads.x, threads.y, threads.z);
  //printf("gridDim: %d, %d, %d\n", blocks.x, blocks.y, blocks.z);


  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "batchnorm1d_backward_cuda", ([&] {
    batchnorm1d_cuda_backward_d_normalized_input_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
    gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
    d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
    d_normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
    );
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_gamma_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                               normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                               );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_gamma.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_beta_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                              d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                              cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                              );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_beta.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_sigma2_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                                mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                                sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                                gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                                d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                                cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                                );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_sigma2.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_mu_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                            input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                            mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                            d_sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                            );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_mu.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_input_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                               input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               d_normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               d_sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               d_mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               d_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                               );
  hipDeviceSynchronize();}));

  return { d_input, d_gamma, d_beta };
}

