#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>
#include "sharedmem.cuh"

#include <vector>

namespace {

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_forward_mean_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> mu_cache) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      mu_cache[n][i] = input[n][i] / batch_size;
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_forward_sigma2_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> sigma2_cache
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      sigma2_cache[n][i] = (input[n][i] - mu[i]) * (input[n][i] - mu[i]) / batch_size;
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_forward_output_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> normalized_input,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> beta,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> output) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      normalized_input[n][i] = (input[n][i] - mu[i]) * rsqrt(sigma2[i] + 1e-5);
      output[n][i] = gamma[i] * normalized_input[n][i] + beta[i];
    }
  }

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_normalized_input_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_normalized_input
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_normalized_input[n][i] = d_output[n][i] * gamma[i];
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_gamma_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> normalized_input,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_gamma_cache
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_gamma_cache[n][i] = d_output[n][i] * normalized_input[n][i];

    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_beta_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_beta_cache
  ) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_beta_cache[n][i] = d_output[n][i];
    }
  }
  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_sigma2_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_sigma2_cache) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_sigma2_cache[n][i] = d_output[n][i] * (input[n][i] - mu[i]) * ((gamma[i] / (-2)) * rsqrt(sigma2[i] + 1e-5) * rsqrt(sigma2[i] + 1e-5) * rsqrt(sigma2[i] + 1e-5));
    }
  }

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_mu_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> gamma,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_output,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_sigma2,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_mu_cache) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_mu_cache[n][i] = (((-2) * d_sigma2[i] * (input[n][i] - mu[i])) / batch_size) + (d_output[n][i] * (-1) * gamma[i] * rsqrt(sigma2[i] + 1e-5));
    }
  }

  template <typename scalar_t>
  __global__ void batchnorm1d_cuda_backward_d_input_kernel(
    const size_t C, const size_t batch_size,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> mu,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> normalized_input,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_normalized_input,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_sigma2,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> d_mu,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> d_input) {

    //batch index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    //channel index
    const int i = threadIdx.y;

    if (n < batch_size) {
      d_input[n][i] = (d_normalized_input[n][i] * rsqrt(sigma2[i] + 1e-5)) + (d_sigma2[i] * ((2 * (input[n][i] - mu[i])) / batch_size) + (d_mu[i] / batch_size));
    }
  }
} // namespace

template <typename scalar_t>
__global__ void reduction_sum1d_kernel(scalar_t* array, scalar_t* sum_output, const int64_t total_num) {

  SharedMemory<scalar_t> shared_memory;
  scalar_t* shared_data = shared_memory.getPointer();
  const auto tid = threadIdx.x;
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;


  if (i < total_num) {
    shared_data[tid] = array[i];
  }
  else {
    shared_data[tid] = 0;
  }

  __syncthreads();

  for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      shared_data[tid] += shared_data[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0) {
    sum_output[blockIdx.x] = shared_data[0];
  }
}

template <typename scalar_t>
scalar_t reduction_sum(
  torch::Tensor array
) {

  const auto total_num = array.numel();
  const auto thread_num = 1024;
  const auto block_num = (total_num + 1024 - 1) / 1024;

  scalar_t* block_sums; // block_sum[-1] represents the total sum of array
  hipMalloc((void**)&block_sums, sizeof(scalar_t) * (1 + block_num));

  reduction_sum1d_kernel<scalar_t> << < block_num, thread_num, thread_num * sizeof(scalar_t) >> > (array.data_ptr<scalar_t>(), block_sums, total_num);
  hipDeviceSynchronize();
  reduction_sum1d_kernel<scalar_t> << <1, thread_num, thread_num * sizeof(scalar_t) >> > (block_sums, block_sums + block_num, block_num);
  hipDeviceSynchronize();


  scalar_t total_sum;
  hipMemcpy(&total_sum, block_sums + block_num, sizeof(scalar_t), hipMemcpyDeviceToHost);
  hipFree(block_sums);


  return total_sum;

}

std::vector<torch::Tensor> batchnorm1d_cuda_forward(
  torch::Tensor input,
  torch::Tensor gamma,
  torch::Tensor beta) {
  const auto batch_size = input.size(0);
  const auto C = input.size(1);

  const int thread_batch_num = int(floor(sqrt(1024 / C)));

  auto mu = torch::zeros_like(gamma);
  auto sigma2 = torch::zeros_like(gamma);
  auto normalized_input = torch::zeros_like(input);
  auto output = torch::zeros_like(input);
  auto cache = torch::empty_like(input);

  const dim3 threads(thread_batch_num, C, 1);
  const dim3 blocks((batch_size + thread_batch_num - 1) / thread_batch_num, 1, 1);



  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "batchnorm1d_forward_cuda", ([&] {
    batchnorm1d_cuda_forward_mean_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
    input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
    cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    mu.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_forward_sigma2_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                             input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    sigma2.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_forward_output_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                             input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                             gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             beta.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                             output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
  hipDeviceSynchronize();
                                                                               }));
  return { input, mu, sigma2, normalized_input, gamma, beta, output };
}

std::vector<torch::Tensor> batchnorm1d_cuda_backward(
  torch::Tensor d_output,
  torch::Tensor input,
  torch::Tensor mu,
  torch::Tensor sigma2,
  torch::Tensor normalized_input,
  torch::Tensor gamma,
  torch::Tensor beta) {
  const auto batch_size = input.size(0);
  const auto C = input.size(1);

  const int thread_batch_num = int(floor(sqrt(1024 / C)));

  auto cache = torch::empty_like(input);

  auto d_input = torch::empty_like(input);
  auto d_gamma = torch::empty_like(gamma);
  auto d_beta = torch::empty_like(beta);

  auto d_normalized_input = torch::empty_like(normalized_input);
  auto d_sigma2 = torch::empty_like(sigma2);
  auto d_mu = torch::empty_like(mu);

  const dim3 threads(thread_batch_num, C, 1);
  const dim3 blocks((batch_size + thread_batch_num - 1) / thread_batch_num, 1, 1);



  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "batchnorm1d_backward_cuda", ([&] {
    batchnorm1d_cuda_backward_d_normalized_input_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
    gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
    d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
    d_normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
    );
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_gamma_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                               normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                               );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_gamma.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_beta_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                              d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                              cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                              );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_beta.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_sigma2_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                                input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                                mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                                sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                                gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                                d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                                cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                                );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_sigma2.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_mu_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                            input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                            mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            gamma.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            d_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                            d_sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                            cache.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                            );
  hipDeviceSynchronize();

  for (int i = 0; i < C; i++) {
    d_mu.index_put_({ i }, reduction_sum<scalar_t>(cache.index({ torch::indexing::Slice(), i }).clone()));
  }
  hipDeviceSynchronize();

  batchnorm1d_cuda_backward_d_input_kernel<scalar_t> << <blocks, threads >> > (C, batch_size,
                                                                               input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               d_normalized_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                                                               d_sigma2.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               d_mu.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                                                               d_input.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>()
                                                                               );
  hipDeviceSynchronize();}));

  return { d_input, d_gamma, d_beta };
}

